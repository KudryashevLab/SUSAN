#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <getopt.h>
#include <unistd.h>

#include "io.h"
#include "aligner.h"
#include "particles.h"
#include "tomogram.h"
#include "reference.h"
#include "aligner_args.h"

void print_data_info(Particles&ptcls,Tomograms&tomos) {
	printf("\t\tAvailable particles:  %d.\n",ptcls.n_ptcl);
	printf("\t\tNumber of classes:    %d.\n",ptcls.n_refs);
	printf("\t\tTomograms available:  %d.\n",tomos.num_tomo);
	printf("\t\tAvailabe projections: %d (max).\n",tomos.num_proj);
	
}

int main(int ac, char** av) {

	ArgsAli::Info info;

	if( ArgsAli::parse_args(info,ac,av) ) {
		ArgsAli::print(info);
                PBarrier barrier(2);
		ParticlesRW ptcls(info.ptcls_in);
		References refs(info.refs_file);
		Tomograms tomos(info.tomo_file);
		print_data_info(ptcls,tomos);
		StackReader stkrdr(&ptcls,&tomos,&barrier);
		AliPool pool(&info,&refs,tomos.num_proj,ptcls.n_ptcl,stkrdr,info.n_threads);
		
                stkrdr.start();
		pool.start();
		
		stkrdr.wait();
		pool.wait();

                ptcls.save(info.ptcls_out);
	}
	else {
		fprintf(stderr,"Error parsing input arguments.\n");
		exit(1);
	}
	
    return 0;
}



